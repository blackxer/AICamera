#include "hip/hip_runtime.h"
#include "ATen/ATen.h"
#include "ATen/InitialTensorOptions.h"
#include "ATen/NativeFunctions.h"
#include "ATen/cuda/HIPContext.h"
#include "c10/util/Exception.h"

#include <THC/THCGeneral.h>
#include <THC/THCThrustAllocator.cuh>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/sequence.h>

#include <algorithm>
#include <cstddef>

namespace at {
namespace native {

Tensor& eye_out_cuda(Tensor& result, int64_t n) {
  return at::native::eye_out_cuda(result, n, /*m=*/-1);
}

Tensor& eye_out_cuda(Tensor& result, int64_t n, int64_t m) {
  AT_CHECK(n >= 0, "n must be greater or equal to 0, got ", n);

  if(m < 0) {
    m = n;
  }

  result.resize_({n, m});
  result.zero_();

  int64_t sz = std::min<int64_t>(n, m);
  int64_t stride = result.stride(0) + result.stride(1);

  Tensor diag = result.as_strided({sz}, {stride});
  diag.fill_(1);
  return result;
}

Tensor empty_cuda(IntList size, const TensorOptions& options) {
  AT_ASSERT(options.backend() == at::Backend::CUDA);
  AT_ASSERT(!options.is_variable());  // is_variable should have been 'unpacked'

  auto* allocator = at::cuda::getCUDADeviceAllocator();
  int64_t nelements = prod_intlist(size);
  auto dtype = options.dtype();
  auto storage_impl = c10::make_intrusive<StorageImpl>(
    dtype,
    nelements,
    allocator->allocate(nelements * dtype.itemsize()),
    allocator,
    /*resizeable=*/true);

  auto tensor = detail::make_tensor<TensorImpl>(storage_impl, CUDATensorId(), false);
  // Default TensorImpl has size [0]
  if (size.size() != 1 || size[0] != 0) {
    tensor.unsafeGetTensorImpl()->set_sizes_contiguous(size);
  }
  return tensor;
}

Tensor& randperm_out_cuda(Tensor& result, int64_t n, Generator* generator) {
  AT_CHECK(n >= 0, "n must be non-negative, got", n);
  AT_CHECK(result.type().scalarTensor(n).defined(),
  "n is too large for result tensor type: '", result.type().toString(), "'");

  result.resize_({n});

  if (result.type().scalarType() == at::ScalarType::Half) {
    auto result_float = at::empty({n}, initialTensorOptions().device(Device(DeviceType::CUDA)));
    result.copy_(randperm_out_cuda(result_float, n, generator));
  } else {
    if (n < 30000) {  // For small inputs, we offload it to CPU instead.
      auto result_cpu = at::empty({n}, result.options().device(kCPU));
      randperm_out(result_cpu, n, generator);
      result.copy_(result_cpu);
    } else {
      // Generate random values for the keys array
      AT_DISPATCH_ALL_TYPES(
        result.type(), "randperm_out_cuda", [&] {
          auto keys = at::empty(result.sizes(), result.options()).random_(generator);

          auto result_data = thrust::device_ptr<scalar_t>(result.data<scalar_t>());
          auto keys_data = thrust::device_ptr<scalar_t>(keys.data<scalar_t>());

          auto state = globalContext().getTHCState();
          THCThrustAllocator thrustAlloc(state);
          auto policy = thrust::cuda::par(thrustAlloc).on(at::cuda::getCurrentCUDAStream());

          thrust::sequence(policy, result_data, result_data + n);

          // Use the sorted order of keys to rearrange the result array
          thrust::sort_by_key(policy, keys_data, keys_data + n, result_data);
        }
      );
    }
  }

  return result;
}

}} // namespace at::native
